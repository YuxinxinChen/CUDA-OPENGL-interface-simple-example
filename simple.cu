#include "hip/hip_runtime.h"
/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

#include <GL/glew.h>
#include <GLFW/glfw3.h>

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

static void error_callback(int error, const char* description)
{
    fprintf(stderr, "Error: %s\n", description);
}

static void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
        glfwSetWindowShouldClose(window, GLFW_TRUE);
}

struct vertexColor
{
	float x,y;
	float r,g,b;
};
	
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

const unsigned int window_width = 512;
const unsigned int window_height = 512;

__global__ void triangle_kernel(vertexColor* pos, unsigned int width, unsigned int height)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    //Calculate the clip coordinates
    float u = 2.0 * x / (float) width - 1;
    float v = 1 - 2.0 * y / (float) height;

    //Calculate a color
    if(u<=1 && u>=-1 && v<=1 && v>=-1)
    {
	if(v <= u+0.5 && v <= -u+0.5 && v >= -0.5)
	{
		pos[x*width+y].x = u;
		pos[x*width+y].y = v;
		pos[x*width+y].r = 255;
		pos[x*width+y].g = 0;
		pos[x*width+y].b = 0;
	}
	else
	{
		pos[x*width+y].x = u;
		pos[x*width+y].y = v;
		pos[x*width+y].r = 0;
		pos[x*width+y].g = 0;
		pos[x*width+y].b = 0;
	}
    }
}

void launch_kernel(vertexColor *pos,  unsigned int width,
                   unsigned int height)
{
    dim3 block(8, 8, 1);
    dim3 grid(width / block.x, height / block.y, 1);
    triangle_kernel<<< grid, block>>>(pos, width, height);
}

void runCuda(struct hipGraphicsResource **vbo_resource)
{
	vertexColor *dptr;
	hipGraphicsMapResources(1, vbo_resource, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource);
	launch_kernel(dptr, window_width, window_height);

	hipGraphicsUnmapResources(1, vbo_resource, 0);
}

void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res, unsigned int vbo_res_flags)
{
	//Create vertex buffer object
	glGenBuffers(1, vbo);
	glBindBuffer(GL_ARRAY_BUFFER, *vbo);
	//Initialize VBO
	unsigned int size = window_width * window_height * sizeof(vertexColor);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	//Register VBO with CUDA
	hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags);
}


int main(void)
{
    //------ InitGL---------------//
    GLFWwindow* window;
    
    glfwSetErrorCallback(error_callback);

    if (!glfwInit())
        exit(EXIT_FAILURE);

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 2);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);

    window = glfwCreateWindow(window_width, window_height, "Simple example", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        exit(EXIT_FAILURE);
    }
    glfwSetKeyCallback(window, key_callback);

    glfwMakeContextCurrent(window);
    glewInit();
    glfwSwapInterval(1);

    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    glViewport(0,0,window_width, window_height);
    //----------InitGL--------------//

    cudaGLSetGLDevice(0);

    createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);
    
    runCuda(&cuda_vbo_resource);

    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(2, GL_FLOAT, sizeof(float)*5, 0);
    glColorPointer(3, GL_FLOAT,sizeof(float)*5, (GLvoid*)(sizeof(float)*2));
    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
    
    while (!glfwWindowShouldClose(window))
    {
        glClear(GL_COLOR_BUFFER_BIT);

        glDrawArrays(GL_POINTS, 0, window_width * window_height);

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    glfwDestroyWindow(window);

    glfwTerminate();
    exit(EXIT_SUCCESS);
}
  
